#include "hip/hip_runtime.h"
//
// Created by jreuter on 07.12.20.
//

#include "toolbox.cuh"
#include <chrono>

__device__ void shellSort(float* array, uint low, uint high) {
    // Using the Ciura, 2001 sequence for best performance
    uint gaps[8] = {1, 4, 10, 23, 57, 132, 301, 701};
    if(low < high) {
        float* subArr = array + low;
        uint n = high - low;
        for (int pos = 7; pos > 0; --pos) {
            uint gap = gaps[pos];
            // Do a gapped insertion sort for this gap size.
            // The first gap elements a[0..gap-1] are already in gapped order
            // keep adding one more element until the entire array is
            // gap sorted
            for (uint i = gap; i < n; i += 1) {
                // add a[i] to the elements that have been gap sorted
                // save a[i] in temp and make a hole at position i
                float temp = subArr[i];

                // shift earlier gap-sorted elements up until the correct
                // location for a[i] is found
                uint j;
                for (j = i; j >= gap && subArr[j - gap] > temp; j -= gap) {
                    subArr[j] = subArr[j - gap];
                }

                // put temp (the original a[i]) in its correct location
                subArr[j] = temp;
            }
        }
    }
}

__global__ void medianFilterKernel(const float* image, int image_stride,
                                   float* result_image, int result_image_stride,
                                   int2 imageDims) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint validValues = 0;
    int cy_bound;

    float buffer[4 * MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE];

    if(x >= MEDIAN_KERNEL_SIZE && x < imageDims.x - MEDIAN_KERNEL_SIZE && y >= MEDIAN_KERNEL_SIZE && y < imageDims.y - MEDIAN_KERNEL_SIZE) {
        // Transfer image pixels to our kernel for median filtering application
        for (int cx = -MEDIAN_KERNEL_SIZE; cx <= MEDIAN_KERNEL_SIZE; ++cx) {
            cy_bound = sqrtf(MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE - cx * cx);
            for (int cy = -cy_bound; cy <= cy_bound; ++cy) {
                buffer[validValues] = image[x + cx + (y + cy) * image_stride];
                ++validValues;
            }
        }
        shellSort(buffer, 0, validValues);
        result_image[x + y * result_image_stride] = buffer[validValues / 2];
    } else {
        result_image[x + y * result_image_stride] = 0.0f;
    }
}

__global__ void medianFilterMaskedKernel(const float* image, int image_stride,
                                         float* result_image, int result_image_stride,
                                         const uchar* mask, int mask_stride,
                                         int2 imageDims) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    uint validValues = 0;
    int cy_bound;

    float buffer[4 * MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE];

    if(x >= MEDIAN_KERNEL_SIZE && x < imageDims.x - MEDIAN_KERNEL_SIZE && y >= MEDIAN_KERNEL_SIZE && y < imageDims.y - MEDIAN_KERNEL_SIZE) {
        if(mask[x + y * mask_stride]) {
            // Transfer image pixels to our kernel for median filtering application
            for (int cx = -MEDIAN_KERNEL_SIZE; cx < MEDIAN_KERNEL_SIZE; ++cx) {
                cy_bound = sqrtf(MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE - cx * cx);
                for (int cy = -cy_bound; cy < cy_bound; ++cy) {
                    if (mask[x + cx + (y + cy) * mask_stride] != 0) {
                        buffer[validValues] = image[x + cx + (y + cy) * image_stride];
                        ++validValues;
                    }
                }
            }
            if (validValues > 1) {
                shellSort(buffer, 0, validValues);
                result_image[x + y * result_image_stride] = buffer[validValues / 2];
            } else if (validValues == 1) {
                result_image[x + y * result_image_stride] = buffer[0];
            } else {
                result_image[x + y * result_image_stride] = 0;
            }
        } else {
            result_image[x + y * result_image_stride] = 0;
        }
    }

}

std::shared_ptr<cv::Mat> PLImg::cuda::filters::callCUDAmedianFilter(const std::shared_ptr<cv::Mat>& image) {
    // Copy the result back to the CPU
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());
    cv::copyMakeBorder(*image, *image, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

    // Error objects
    hipError_t err;

    uint numberOfChunks = 1;
    ulong freeMem;
    err = hipMemGetInfo(&freeMem, nullptr);
    if(err != hipSuccess) {
        std::cerr << "Could not get free memory! \n";
        std::cerr << hipGetErrorName(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    if(double(image->total()) * image->elemSize() * 2.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 2.1 / double(freeMem)) / log(4))));
    }
    uint chunksPerDim = fmax(1, sqrtf(numberOfChunks));

    float* deviceImage, *deviceResult;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nResStep;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;

    int2 realImageDims = {image->cols - 2 * MEDIAN_KERNEL_SIZE, image->rows - 2 * MEDIAN_KERNEL_SIZE};
    int2 subImageDims;

    cv::Mat subImage, subResult, croppedImage;
    for(uint it = 0; it < numberOfChunks; ++it) {
        // Calculate image boarders
        xMin = (it % chunksPerDim) * realImageDims.x / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * realImageDims.x / chunksPerDim, realImageDims.x);
        yMin = (it / chunksPerDim) * realImageDims.y / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * realImageDims.y / chunksPerDim, realImageDims.y);

        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);
        cv::copyMakeBorder(subResult, subResult, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

        err = hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for original transmittance \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nSrcStep = subImage.cols;

        err = hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for resulting image \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        err = hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Apply median filter
        auto start = std::chrono::high_resolution_clock::now();
        subImageDims = {subImage.cols, subImage.rows};
        threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
        numBlocks = dim3(subImageDims.x / threadsPerBlock.x, subImageDims.y / threadsPerBlock.y);
        // Run median filter
        medianFilterKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                           deviceResult, nResStep,
                                                           subImageDims);
        hipDeviceSynchronize();
        auto finish = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = finish - start;
        std::cout << "Elapsed time: " << elapsed.count() << " s\n";

        err = hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from device to host \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Free reserved memory
        hipFree(deviceImage);
        hipFree(deviceResult);     

        cv::Rect srcRect = cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, subResult.cols - 2*MEDIAN_KERNEL_SIZE, subResult.rows - 2*MEDIAN_KERNEL_SIZE);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);

        subResult(srcRect).copyTo(result(dstRect));
    }
    croppedImage = cv::Mat(*image, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, image->cols - 2*MEDIAN_KERNEL_SIZE, image->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*image);
    return std::make_shared<cv::Mat>(result);
}

std::shared_ptr<cv::Mat> PLImg::cuda::filters::callCUDAmedianFilterMasked(const std::shared_ptr<cv::Mat>& image, const std::shared_ptr<cv::Mat>& mask) {
    // Copy the result back to the CPU
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());
    cv::copyMakeBorder(*image, *image, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);
    cv::copyMakeBorder(*mask, *mask, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

    // Error objects
    hipError_t err;

    uint numberOfChunks = 1;
    ulong freeMem;
    err = hipMemGetInfo(&freeMem, nullptr);
    if(err != hipSuccess) {
        std::cerr << "Could not get free memory! \n";
        std::cerr << hipGetErrorName(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    if(double(image->total()) * image->elemSize() * 3.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 3.1 / double(freeMem)) / log(4))));
    }
    uint chunksPerDim = fmax(1, sqrtf(numberOfChunks));

    float* deviceImage, *deviceResult;
    uchar* deviceMask;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nMaskStep, nResStep;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;

    int2 realImageDims = {image->cols - 2 * MEDIAN_KERNEL_SIZE, image->rows - 2 * MEDIAN_KERNEL_SIZE};
    int2 subImageDims;

    cv::Mat subImage, subMask, subResult, croppedImage;
    for(uint it = 0; it < numberOfChunks; ++it) {
        // Calculate image boarders
        xMin = (it % chunksPerDim) * realImageDims.x / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * realImageDims.x / chunksPerDim, realImageDims.x);
        yMin = (it / chunksPerDim) * realImageDims.y / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * realImageDims.y / chunksPerDim, realImageDims.y);

        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(*mask, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subMask);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);
        cv::copyMakeBorder(subResult, subResult, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

        err = hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for original transmittance \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nSrcStep = subImage.cols;

        err = hipMalloc((void **) &deviceMask, subMask.total() * subMask.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for mask \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nMaskStep = subMask.cols;

        err = hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for resulting image \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        err = hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(deviceMask, subMask.data, subMask.total() * subMask.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy mask from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Apply median filter
        subImageDims = {subImage.cols, subImage.rows};
        threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
        numBlocks = dim3(subImageDims.x / threadsPerBlock.x, subImageDims.y / threadsPerBlock.y);
        // Run median filter
        medianFilterMaskedKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                                 deviceResult, nResStep,
                                                                 deviceMask, nMaskStep,
                                                                 subImageDims);

        err = hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from device to host \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Free reserved memory
        hipFree(deviceImage);
        hipFree(deviceResult);
        hipFree(deviceMask);

        cv::Rect srcRect = cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, subResult.cols - 2*MEDIAN_KERNEL_SIZE, subResult.rows - 2*MEDIAN_KERNEL_SIZE);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);

        subResult(srcRect).copyTo(result(dstRect));
    }
    croppedImage = cv::Mat(*image, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, image->cols - 2*MEDIAN_KERNEL_SIZE, image->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*image);
    croppedImage = cv::Mat(*mask, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, mask->cols - 2*MEDIAN_KERNEL_SIZE, mask->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*mask);
    return std::make_shared<cv::Mat>(result);
}


