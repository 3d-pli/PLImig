#include "hip/hip_runtime.h"
//
// Created by jreuter on 07.12.20.
//

#include "toolbox.cuh"
#include <chrono>

__device__ void shellSort(float* array, uint low, uint high) {
    // Using the Ciura, 2001 sequence for best performance
    uint gaps[8] = {1, 4, 10, 23, 57, 132, 301, 701};
    if(low < high) {
        float* subArr = array + low;
        uint n = high - low;
        for (int pos = 7; pos > 0; --pos) {
            uint gap = gaps[pos];
            // Do a gapped insertion sort for this gap size.
            // The first gap elements a[0..gap-1] are already in gapped order
            // keep adding one more element until the entire array is
            // gap sorted
            for (uint i = gap; i < n; i += 1) {
                // add a[i] to the elements that have been gap sorted
                // save a[i] in temp and make a hole at position i
                float temp = subArr[i];

                // shift earlier gap-sorted elements up until the correct
                // location for a[i] is found
                uint j;
                for (j = i; j >= gap && subArr[j - gap] > temp; j -= gap) {
                    subArr[j] = subArr[j - gap];
                }

                // put temp (the original a[i]) in its correct location
                subArr[j] = temp;
            }
        }
    }
}

__global__ void medianFilterKernel(const float* image, int image_stride, int2 image_offset,
                                   float* result_image, int result_image_stride, int2 result_offset,
                                   int2 roi, int2 anchor) {
    uint thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    uint thread_y = blockIdx.y * blockDim.y + threadIdx.y;

    uint x = thread_x - anchor.x + image_offset.x;
    uint y = thread_y - anchor.y + image_offset.y;
    uint rx = thread_x - anchor.x + result_offset.x;
    uint ry = thread_y - anchor.y + result_offset.y;

    uint validValues = 0;
    int cy_bound;

    float buffer[4 * KERNEL_SIZE * KERNEL_SIZE];

    if(x > KERNEL_SIZE && x < roi.x && y > KERNEL_SIZE && y < roi.y) {
        // Transfer image pixels to our kernel for median filtering application
        for (int cx = -KERNEL_SIZE; cx <= KERNEL_SIZE; ++cx) {
            cy_bound = sqrtf(KERNEL_SIZE * KERNEL_SIZE - cx * cx);
            for (int cy = -cy_bound; cy <= cy_bound; ++cy) {
                buffer[validValues] = image[x + cx + (y + cy) * image_stride];
                ++validValues;
            }
        }
        shellSort(buffer, 0, validValues);
        result_image[rx + ry * result_image_stride] = buffer[validValues / 2];
    } else {
        result_image[rx + ry * result_image_stride] = 0;
    }
    //printf("Valid values: %d\n", validValues);
}

__global__ void medianFilterMaskedKernel(const float* image, int image_stride, int2 image_offset,
                                         float* result_image, int result_image_stride, int2 result_offset,
                                         const uchar* mask, int mask_stride, int2 mask_offset,
                                         int2 roi, int2 anchor) {
    uint thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    uint thread_y = blockIdx.y * blockDim.y + threadIdx.y;

    uint x = thread_x - anchor.x + image_offset.x;
    uint y = thread_y - anchor.y + image_offset.y;
    uint rx = thread_x - anchor.x + result_offset.x;
    uint ry = thread_y - anchor.y + result_offset.y;
    uint mx = thread_x - anchor.x + mask_offset.x;
    uint my = thread_y - anchor.y + mask_offset.y;

    float buffer[KERNEL_SIZE * KERNEL_SIZE];
    uint validValues = 0;

    if(x > KERNEL_SIZE && x < roi.x && y > KERNEL_SIZE && y < roi.y) {
        if(mask[mx + my * mask_stride]) {
            // Transfer image pixels to our kernel for median filtering application
            for (uint cx = 0; cx < KERNEL_SIZE; ++cx) {
                for (uint cy = 0; cy < KERNEL_SIZE; ++cy) {
                    if (mask[mx + cx + (my + cy) * mask_stride] != 0) {
                        buffer[validValues] = image[x + cx + (y + cy) * image_stride];
                        ++validValues;
                    }
                }
            }
            if (validValues > 1) {
                shellSort(buffer, 0, validValues);
                result_image[rx + ry * result_image_stride] = buffer[validValues / 2];
            } else if (validValues == 1) {
                result_image[rx + ry * result_image_stride] = buffer[0];
            } else {
                result_image[rx + ry * result_image_stride] = 0;
            }
        } else {
            result_image[rx + ry * result_image_stride] = 0;
        }
    }

}

std::shared_ptr<cv::Mat> PLImg::cuda::filters::callCUDAmedianFilter(const std::shared_ptr<cv::Mat>& image) {
    // Copy the result back to the CPU
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());

    // Error objects
    hipError_t err;

    uint numberOfChunks = 1;
    ulong freeMem;
    err = hipMemGetInfo(&freeMem, nullptr);
    if(err != hipSuccess) {
        std::cerr << "Could not get free memory! \n";
        std::cerr << hipGetErrorName(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    if(double(image->total()) * image->elemSize() * 2.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 2.1 / double(freeMem)) / log(4))));
    }
    uint chunksPerDim = fmax(1, numberOfChunks/2);

    float* deviceImage, *deviceResult;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nResStep;
    // Apply median filter
    // Set size where median filter will be applied
    int2 roi;
    // Median kernel
    int2 anchor = {KERNEL_SIZE, KERNEL_SIZE};
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    int2 pSrcOffset = {KERNEL_SIZE, KERNEL_SIZE};
    int2 pResultOffset = {KERNEL_SIZE, KERNEL_SIZE};
    dim3 threadsPerBlock, numBlocks;

    cv::Mat subImage, subResult, croppedImage;
    for(uint it = 0; it < numberOfChunks; ++it) {
        // Calculate image boarders
        xMin = (it % chunksPerDim) * image->cols / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * image->cols / chunksPerDim, image->cols);
        yMin = (it / chunksPerDim) * image->rows / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * image->rows / chunksPerDim, image->rows);

        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);

        cv::copyMakeBorder(subImage, subImage, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, cv::BORDER_REPLICATE);
        cv::copyMakeBorder(subResult, subResult, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, cv::BORDER_REPLICATE);

        err = hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for original transmittance \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nSrcStep = subImage.cols;

        err = hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for resulting image \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        err = hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Apply median filter
        auto start = std::chrono::high_resolution_clock::now();
        roi = {subImage.cols - 2 * KERNEL_SIZE, subImage.rows - 2 * KERNEL_SIZE};
        threadsPerBlock = dim3(NUM_THREADS, NUM_THREADS);
        numBlocks = dim3(roi.x / threadsPerBlock.x, roi.y / threadsPerBlock.y);
        // Run median filter
        medianFilterKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep, pSrcOffset,
                                                           deviceResult, nResStep, pResultOffset,
                                                           roi, anchor);
        hipDeviceSynchronize();
        auto finish = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = finish - start;
        std::cout << "Elapsed time: " << elapsed.count() << " s\n";

        err = hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from device to host \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Free reserved memory
        hipFree(deviceImage);
        hipFree(deviceResult);

        cv::Rect srcRect = cv::Rect(KERNEL_SIZE, KERNEL_SIZE, subResult.cols - 2*KERNEL_SIZE, subResult.rows - 2*KERNEL_SIZE);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);

        subResult(srcRect).copyTo(result(dstRect));
    }
    return std::make_shared<cv::Mat>(result);
}

std::shared_ptr<cv::Mat> PLImg::cuda::filters::callCUDAmedianFilterMasked(const std::shared_ptr<cv::Mat>& image, const std::shared_ptr<cv::Mat>& mask) {
    // Copy the result back to the CPU
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());

    // Error objects
    hipError_t err;

    uint numberOfChunks = 1;
    ulong freeMem;
    err = hipMemGetInfo(&freeMem, nullptr);
    if(err != hipSuccess) {
        std::cerr << "Could not get free memory! \n";
        std::cerr << hipGetErrorName(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    if(double(image->total()) * image->elemSize() * 3.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 3.1 / double(freeMem)) / log(4))));
    }
    uint chunksPerDim = fmax(1, numberOfChunks/2);

    float* deviceImage, *deviceResult;
    uchar* deviceMask;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nMaskStep, nResStep;
    // Apply median filter
    // Set size where median filter will be applied
    int2 roi;
    // Median kernel
    int2 anchor = {KERNEL_SIZE / 2, KERNEL_SIZE / 2};
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    int2 pSrcOffset = {KERNEL_SIZE, KERNEL_SIZE};
    int2 pResultOffset = {KERNEL_SIZE, KERNEL_SIZE};
    int2 pMaskOffset = {KERNEL_SIZE, KERNEL_SIZE};
    dim3 threadsPerBlock, numBlocks;

    cv::Mat subImage, subMask, subResult, croppedImage;
    for(uint it = 0; it < numberOfChunks; ++it) {
        // Calculate image boarders
        xMin = (it % chunksPerDim) * image->cols / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * image->cols / chunksPerDim, image->cols);
        yMin = (it / chunksPerDim) * image->rows / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * image->rows / chunksPerDim, image->rows);

        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(*mask, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subMask);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);

        cv::copyMakeBorder(subImage, subImage, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, cv::BORDER_REPLICATE);
        cv::copyMakeBorder(subResult, subResult, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, cv::BORDER_REPLICATE);
        cv::copyMakeBorder(subMask, subMask, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, KERNEL_SIZE, cv::BORDER_REPLICATE);

        err = hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for original transmittance \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nSrcStep = subImage.cols;

        err = hipMalloc((void **) &deviceMask, subMask.total() * subMask.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for mask \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nMaskStep = subMask.cols;

        err = hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for resulting image \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        err = hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(deviceMask, subMask.data, subMask.total() * subMask.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy mask from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Apply median filter
        roi = {subImage.cols - 2 * KERNEL_SIZE, subImage.rows - 2 * KERNEL_SIZE};
        threadsPerBlock = dim3(NUM_THREADS, NUM_THREADS);
        numBlocks = dim3(roi.x / threadsPerBlock.x, roi.y / threadsPerBlock.y);
        // Run median filter
        medianFilterMaskedKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep, pSrcOffset,
                                                                 deviceResult, nResStep, pResultOffset,
                                                                 deviceMask, nMaskStep, pMaskOffset,
                                                                 roi, anchor);

        err = hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from device to host \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Free reserved memory
        hipFree(deviceImage);
        hipFree(deviceResult);
        hipFree(deviceMask);

        cv::Rect srcRect = cv::Rect(KERNEL_SIZE, KERNEL_SIZE, subResult.cols - 2*KERNEL_SIZE, subResult.rows - 2*KERNEL_SIZE);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);

        subResult(srcRect).copyTo(result(dstRect));
    }
    return std::make_shared<cv::Mat>(result);
}


