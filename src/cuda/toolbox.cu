#include "hip/hip_runtime.h"
//
// Created by jreuter on 07.12.20.
//

#include "toolbox.cuh"

__device__ void shellSort(float* array, uint low, uint high) {
    // Using the Ciura, 2001 sequence for best performance
    uint gaps[8] = {1, 4, 10, 23, 57, 132, 301, 701};
    if(low < high) {
        float* subArr = array + low;
        uint n = high - low;
        for (int pos = 7; pos > 0; --pos) {
            uint gap = gaps[pos];
            // Do a gapped insertion sort for this gap size.
            // The first gap elements a[0..gap-1] are already in gapped order
            // keep adding one more element until the entire array is
            // gap sorted
            for (uint i = gap; i < n; i += 1) {
                // add a[i] to the elements that have been gap sorted
                // save a[i] in temp and make a hole at position i
                float temp = subArr[i];

                // shift earlier gap-sorted elements up until the correct
                // location for a[i] is found
                uint j;
                for (j = i; j >= gap && subArr[j - gap] > temp; j -= gap) {
                    subArr[j] = subArr[j - gap];
                }

                // put temp (the original a[i]) in its correct location
                subArr[j] = temp;
            }
        }
    }
}

__global__ void medianFilterKernel(const float* image, int image_stride,
                                   float* result_image, int result_image_stride,
                                   int2 imageDims) {
    // Calculate actual position in image based on thread number and block number
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    // The valid values will be counted to ensure that the median will be calculated correctly
    uint validValues = 0;
    int cy_bound;
    // Median filter buffer
    float buffer[4 * MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE];

    // Only try to calculate the median of pixels within the non-padded image
    if(x >= MEDIAN_KERNEL_SIZE && x < imageDims.x - MEDIAN_KERNEL_SIZE && y >= MEDIAN_KERNEL_SIZE && y < imageDims.y - MEDIAN_KERNEL_SIZE) {
        // Transfer image pixels to our kernel for median filtering application
        for (int cx = -MEDIAN_KERNEL_SIZE; cx <= MEDIAN_KERNEL_SIZE; ++cx) {
            // The median filter kernel is round. Therefore calculate the valid y-positions based on our x-position in the kernel
            cy_bound = sqrtf(MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE - cx * cx);
            for (int cy = -cy_bound; cy <= cy_bound; ++cy) {
                // Save values in buffer
                buffer[validValues] = image[x + cx + (y + cy) * image_stride];
                ++validValues;
            }
        }
        shellSort(buffer, 0, validValues);
        // Get middle value as our median
        result_image[x + y * result_image_stride] = buffer[validValues / 2];
    }
}

__global__ void medianFilterMaskedKernel(const float* image, int image_stride,
                                         float* result_image, int result_image_stride,
                                         const uchar* mask, int mask_stride,
                                         int2 imageDims) {
    // Calculate actual position in image based on thread number and block number
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;
    // The valid values will be counted to ensure that the median will be calculated correctly
    uint validValues = 0;
    int cy_bound;
    // Median filter buffer
    float buffer[4 * MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE];

    // Only try to calculate the median of pixels within the non-padded image
    if(x > MEDIAN_KERNEL_SIZE && x < imageDims.x - MEDIAN_KERNEL_SIZE && y > MEDIAN_KERNEL_SIZE && y < imageDims.y - MEDIAN_KERNEL_SIZE) {
        // Check if the current pixel is on the mask
        if(mask[x + y * mask_stride]) {
            // Transfer image pixels to our kernel for median filtering application
            for (int cx = -MEDIAN_KERNEL_SIZE; cx < MEDIAN_KERNEL_SIZE; ++cx) {
                // The median filter kernel is round. Therefore calculate the valid y-positions based on our x-position in the kernel
                cy_bound = sqrtf(MEDIAN_KERNEL_SIZE * MEDIAN_KERNEL_SIZE - cx * cx);
                for (int cy = -cy_bound; cy < cy_bound; ++cy) {
                    // Check if the pixel on our kernel is on the mask
                    if (mask[x + cx + (y + cy) * mask_stride] != 0) {
                        // Save values in buffer
                        buffer[validValues] = image[x + cx + (y + cy) * image_stride];
                        ++validValues;
                    }
                }
            }
            // Depending on the number of valid values, calculate the median, save the pixel value itself or save zero
            if (validValues > 1) {
                shellSort(buffer, 0, validValues);
                result_image[x + y * result_image_stride] = buffer[validValues / 2];
            } else if (validValues == 1) {
                result_image[x + y * result_image_stride] = buffer[0];
            } else {
                result_image[x + y * result_image_stride] = 0;
            }
        }
    }

}

std::shared_ptr<cv::Mat> PLImg::cuda::filters::callCUDAmedianFilter(const std::shared_ptr<cv::Mat>& image) {
    // Create a result image with the same dimensions as our input image
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());
    // Expand borders of input image inplace to ensure that the median algorithm can run correcly
    cv::copyMakeBorder(*image, *image, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

    // Error objects
    hipError_t err;

    // The image might be too large to be saved completely in the video memory.
    // Therefore chunks will be used if the amount of memory is too small.
    uint numberOfChunks = 1;
    // Check the free video memory
    ulong freeMem;
    err = hipMemGetInfo(&freeMem, nullptr);
    if(err != hipSuccess) {
        std::cerr << "Could not get free memory! \n";
        std::cerr << hipGetErrorName(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    // If the total free memory is smaller than the estimated amount of memory, calculate the number of chunks
    // with the power of four (1, 4, 16, 256, 1024, ...)
    if(double(image->total()) * image->elemSize() * 2.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 2.1 / double(freeMem)) / log(4))));
    }
    // Each dimensions will get the same number of chunks. Calculate them by using the square root.
    uint chunksPerDim = fmax(1, sqrtf(numberOfChunks));

    float* deviceImage, *deviceResult;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nResStep;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;

    // We've increased the image dimensions earlier. Save the original image dimensions for further calculations.
    int2 realImageDims = {image->cols - 2 * MEDIAN_KERNEL_SIZE, image->rows - 2 * MEDIAN_KERNEL_SIZE};
    int2 subImageDims;

    cv::Mat subImage, subResult, croppedImage;
    // For each chunk
    for(uint it = 0; it < numberOfChunks; ++it) {
        std::cout << "\rCurrent chunk: " << it+1 << "/" << numberOfChunks;
        std::flush(std::cout);
        // Calculate image boarders
        xMin = (it % chunksPerDim) * realImageDims.x / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * realImageDims.x / chunksPerDim, realImageDims.x);
        yMin = (it / chunksPerDim) * realImageDims.y / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * realImageDims.y / chunksPerDim, realImageDims.y);

        // Get chunk of our image and result. Apply padding to the result to ensure that the median filter will run correctly.
        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);
        cv::copyMakeBorder(subResult, subResult, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

        // Allocate GPU memory for the original image and its result
        err = hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for original transmittance \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nSrcStep = subImage.cols;

        err = hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for resulting image \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        err = hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Apply median filter
        subImageDims = {subImage.cols, subImage.rows};
        threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
        numBlocks = dim3(ceil(float(subImageDims.x) / threadsPerBlock.x), ceil(float(subImageDims.y) / threadsPerBlock.y));
        // Run median filter
        medianFilterKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                           deviceResult, nResStep,
                                                           subImageDims);

        // Copy result from GPU back to CPU
        err = hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from device to host \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Free reserved memory
        hipFree(deviceImage);
        hipFree(deviceResult);     

        // Calculate the range where the median filter was applied and where the chunk will be placed.
        cv::Rect srcRect = cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, xMax - xMin, yMax - yMin);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);
        subResult(srcRect).copyTo(result(dstRect));
    }
    // Fix output after \r
    std::cout << std::endl;
    // Revert the padding of the original image
    croppedImage = cv::Mat(*image, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, image->cols - 2*MEDIAN_KERNEL_SIZE, image->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*image);

    // Return resulting median filtered image
    return std::make_shared<cv::Mat>(result);
}

std::shared_ptr<cv::Mat> PLImg::cuda::filters::callCUDAmedianFilterMasked(const std::shared_ptr<cv::Mat>& image, const std::shared_ptr<cv::Mat>& mask) {
    // Copy the result back to the CPU
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());
    cv::copyMakeBorder(*image, *image, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);
    cv::copyMakeBorder(*mask, *mask, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

    // Error objects
    hipError_t err;

    // The image might be too large to be saved completely in the video memory.
    // Therefore chunks will be used if the amount of memory is too small.
    uint numberOfChunks = 1;
    ulong freeMem;
    // Check the free video memory
    err = hipMemGetInfo(&freeMem, nullptr);
    if(err != hipSuccess) {
        std::cerr << "Could not get free memory! \n";
        std::cerr << hipGetErrorName(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    // If the total free memory is smaller than the estimated amount of memory, calculate the number of chunks
    // with the power of four (1, 4, 16, 256, 1024, ...)
    if(double(image->total()) * image->elemSize() * 3.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 3.1 / double(freeMem)) / log(4))));
    }
    // Each dimensions will get the same number of chunks. Calculate them by using the square root.
    uint chunksPerDim = fmax(1, sqrtf(numberOfChunks));

    float* deviceImage, *deviceResult;
    uchar* deviceMask;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nMaskStep, nResStep;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;

    // We've increased the image dimensions earlier. Save the original image dimensions for further calculations.
    int2 realImageDims = {image->cols - 2 * MEDIAN_KERNEL_SIZE, image->rows - 2 * MEDIAN_KERNEL_SIZE};
    int2 subImageDims;

    cv::Mat subImage, subMask, subResult, croppedImage;
    for(uint it = 0; it < numberOfChunks; ++it) {
        std::cout << "\rCurrent chunk: " << it+1 << "/" << numberOfChunks;
        std::flush(std::cout);
        // Calculate image boarders
        xMin = (it % chunksPerDim) * realImageDims.x / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * realImageDims.x / chunksPerDim, realImageDims.x);
        yMin = (it / chunksPerDim) * realImageDims.y / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * realImageDims.y / chunksPerDim, realImageDims.y);

        // Get chunk of our image, mask and result. Apply padding to the result to ensure that the median filter will run correctly.
        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(*mask, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subMask);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);
        cv::copyMakeBorder(subResult, subResult, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

        // Allocate GPU memory for the original image, mask and its result
        err = hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for original transmittance \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nSrcStep = subImage.cols;

        err = hipMalloc((void **) &deviceMask, subMask.total() * subMask.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for mask \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nMaskStep = subMask.cols;

        err = hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize());
        if (err != hipSuccess) {
            std::cerr << "Could not allocate enough memory for resulting image \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        err = hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(deviceMask, subMask.data, subMask.total() * subMask.elemSize(), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Could not copy mask from host to device \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Apply median filter
        subImageDims = {subImage.cols, subImage.rows};
        threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
        numBlocks = dim3(ceil(float(subImageDims.x) / threadsPerBlock.x), ceil(float(subImageDims.y) / threadsPerBlock.y));
        // Run median filter
        medianFilterMaskedKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                                 deviceResult, nResStep,
                                                                 deviceMask, nMaskStep,
                                                                 subImageDims);

        err = hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Could not copy image from device to host \n";
            std::cerr << hipGetErrorName(err) << std::endl;
            exit(EXIT_FAILURE);
        }

        // Free reserved memory
        hipFree(deviceImage);
        hipFree(deviceResult);
        hipFree(deviceMask);

        cv::Rect srcRect = cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, subResult.cols - 2*MEDIAN_KERNEL_SIZE, subResult.rows - 2*MEDIAN_KERNEL_SIZE);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);

        subResult(srcRect).copyTo(result(dstRect));
    }
    // Fix output after \r
    std::cout << std::endl;

    croppedImage = cv::Mat(*image, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, image->cols - 2*MEDIAN_KERNEL_SIZE, image->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*image);
    croppedImage = cv::Mat(*mask, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, mask->cols - 2*MEDIAN_KERNEL_SIZE, mask->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*mask);
    return std::make_shared<cv::Mat>(result);
}


