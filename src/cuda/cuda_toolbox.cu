#include "hip/hip_runtime.h"
/*
    MIT License

    Copyright (c) 2021 Forschungszentrum Jülich / Jan André Reuter.

    Permission is hereby granted, free of charge, to any person obtaining a copy
    of this software and associated documentation files (the "Software"), to deal
    in the Software without restriction, including without limitation the rights
    to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
    copies of the Software, and to permit persons to whom the Software is
    furnished to do so, subject to the following conditions:

    The above copyright notice and this permission notice shall be included in all
    copies or substantial portions of the Software.

    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
    AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
    SOFTWARE.
 */

#include "cuda/cuda_toolbox.h"

cv::Mat PLImg::cuda::raw::labeling::CUDAConnectedComponents(const cv::Mat& image, uint* maxLabelNumber) {
    // Prepare image for CUDA kernel
    cv::Mat kernelImage;
    // 1. Convert it to 8 bit unsigned integer values
    image.convertTo(kernelImage, CV_8UC1);
    // 2. Check if the image needs padding to allow the execution of our CUDA kernel
    uint heightPadding = CUDA_KERNEL_NUM_THREADS - kernelImage.rows % CUDA_KERNEL_NUM_THREADS;
    uint widthPadding = CUDA_KERNEL_NUM_THREADS - kernelImage.cols % CUDA_KERNEL_NUM_THREADS;
    cv::copyMakeBorder(kernelImage, kernelImage, heightPadding, 0, widthPadding, 0, cv::BORDER_CONSTANT, 0);

    // Create output resulting image for our needs
    cv::Mat result = cv::Mat(kernelImage.rows, kernelImage.cols, CV_32SC1);

    uchar* deviceImage;
    uint* deviceMask;
    bool* deviceChangeOccured;
    bool changeOccured;

    CHECK_CUDA(hipMalloc(&deviceImage, kernelImage.total() * sizeof(uchar)));
    CHECK_CUDA(hipMemcpy(deviceImage, kernelImage.data, kernelImage.total() * sizeof(uchar), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMalloc(&deviceMask, kernelImage.total() * sizeof(uint)));
    CHECK_CUDA(hipMalloc(&deviceChangeOccured, sizeof(bool)));

    dim3 threadsPerBlock, numBlocks;
    threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
    numBlocks = dim3(ceil(float(kernelImage.cols) / threadsPerBlock.x), ceil(float(kernelImage.rows) / threadsPerBlock.y));

    connectedComponentsInitializeMask<<<numBlocks, threadsPerBlock>>>(deviceImage, kernelImage.cols, deviceMask, kernelImage.cols, kernelImage.cols);
    CHECK_CUDA(hipFree(deviceImage));
    do {
        CHECK_CUDA(hipMemset(deviceChangeOccured, false, sizeof(bool)));
        connectedComponentsIteration<<<numBlocks, threadsPerBlock>>>(deviceMask, kernelImage.cols, {kernelImage.cols, kernelImage.rows},
                                                                     deviceChangeOccured);
        CHECK_CUDA(hipMemcpy(&changeOccured, deviceChangeOccured, sizeof(bool), hipMemcpyDeviceToHost));
    } while(changeOccured);
    CHECK_CUDA(hipFree(deviceChangeOccured));

    uint* deviceUniqueMask;
    CHECK_CUDA(hipMalloc(&deviceUniqueMask, kernelImage.total() * sizeof(uint)));
    CHECK_CUDA(hipMemcpy(deviceUniqueMask, deviceMask, kernelImage.total() * sizeof(uint), hipMemcpyDeviceToDevice));
    thrust::sort(thrust::device, deviceUniqueMask, deviceUniqueMask + kernelImage.total());
    uint* deviceMaxUniqueLabel = thrust::unique(thrust::device, deviceUniqueMask, deviceUniqueMask + kernelImage.total());

    uint distance = thrust::distance(deviceUniqueMask, deviceMaxUniqueLabel);
    connectedComponentsReduceComponents<<<numBlocks, threadsPerBlock>>>(deviceMask, kernelImage.cols,
                                                                        deviceUniqueMask,
                                                                        distance);
    CHECK_CUDA(hipFree(deviceUniqueMask));

    uint* deviceMaxLabel = thrust::max_element(thrust::device, deviceMask, deviceMask + kernelImage.total());
    CHECK_CUDA(hipMemcpy(maxLabelNumber, deviceMaxLabel, sizeof(uint), hipMemcpyDeviceToHost));

    // Copy result from GPU back to CPU
    CHECK_CUDA(hipMemcpy(result.data, deviceMask, kernelImage.total() * sizeof(uint), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(deviceMask));
    CHECK_CUDA(hipDeviceSynchronize());

    cv::Mat croppedImage = cv::Mat(result, cv::Rect(widthPadding, heightPadding, result.cols - widthPadding, result.rows - heightPadding));
    croppedImage.copyTo(result);

    return result;
}

cv::Mat PLImg::cuda::raw::labeling::CUDAConnectedComponentsUF(const cv::Mat &image, uint *maxLabelNumber) {
    // Prepare image for CUDA kernel
    cv::Mat kernelImage;
    // 1. Convert it to 8 bit unsigned integer values
    image.convertTo(kernelImage, CV_8UC1);
    // 2. Check if the image needs padding to allow the execution of our CUDA kernel
    uint heightPadding = CUDA_KERNEL_NUM_THREADS - kernelImage.rows % CUDA_KERNEL_NUM_THREADS;
    uint widthPadding = CUDA_KERNEL_NUM_THREADS - kernelImage.cols % CUDA_KERNEL_NUM_THREADS;
    cv::copyMakeBorder(kernelImage, kernelImage, heightPadding, 0, widthPadding, 0, cv::BORDER_CONSTANT, 0);

    // Create output resulting image for our needs
    cv::Mat result = cv::Mat(kernelImage.rows, kernelImage.cols, CV_32SC1);
    uint* deviceMask;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    hipArray* imageArray;
    CHECK_CUDA(hipMallocArray(&imageArray, &channelDesc, kernelImage.cols, kernelImage.rows));
    CHECK_CUDA(hipMemcpy2DToArray(imageArray, 0, 0, kernelImage.data, kernelImage.cols * sizeof(uchar), kernelImage.cols * sizeof(uchar), kernelImage.rows, hipMemcpyHostToDevice));

    // Step 1. Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = imageArray;
    // Step 2. Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Step 3: Create texture object
    hipTextureObject_t texObj = 0;
    CHECK_CUDA(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));
    CHECK_CUDA(hipMalloc(&deviceMask, kernelImage.total() * sizeof(uint)));

    // Define CUDA kernel parameters
    dim3 threadsPerBlock, numBlocks;
    threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
    numBlocks = dim3(ceil(float(kernelImage.cols) / threadsPerBlock.x), ceil(float(kernelImage.rows) / threadsPerBlock.y));

    // First step. Do local connected components on each block
    connectedComponentsUFLocalMerge<<<numBlocks, threadsPerBlock>>>(texObj, kernelImage.cols, kernelImage.rows, deviceMask, kernelImage.cols);
    CHECK_CUDA(hipDeviceSynchronize());
    // Second step. Fix lines between each block.
    connectedComponentsUFGlobalMerge<<<numBlocks, threadsPerBlock>>>(texObj, kernelImage.cols, kernelImage.rows, deviceMask, kernelImage.cols);
    CHECK_CUDA(hipDeviceSynchronize());
    // Third step. Fix paths which might be wrong after the global merge
    connectedComponentsUFPathCompression<<<numBlocks, threadsPerBlock>>>(texObj, kernelImage.cols, kernelImage.rows, deviceMask, kernelImage.cols);
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipDestroyTextureObject(texObj));
    CHECK_CUDA(hipFreeArray(imageArray));

    // Fourth step. Reduce label numbers to reasonable numbers.
    uint* deviceUniqueMask;
    CHECK_CUDA(hipMalloc(&deviceUniqueMask, kernelImage.total() * sizeof(uint)));
    CHECK_CUDA(hipMemcpy(deviceUniqueMask, deviceMask, kernelImage.total() * sizeof(uint), hipMemcpyDeviceToDevice));
    thrust::sort(thrust::device, deviceUniqueMask, deviceUniqueMask + kernelImage.total());
    uint* deviceMaxUniqueLabel = thrust::unique(thrust::device, deviceUniqueMask, deviceUniqueMask + kernelImage.total());
    // Save the new maximum label as a return value for the user
    uint distance = thrust::distance(deviceUniqueMask, deviceMaxUniqueLabel);
    if(maxLabelNumber) {
        *maxLabelNumber = distance;
    }
    // Reduce numbers in label image to low numbers for following algorithms
    connectedComponentsReduceComponents<<<numBlocks, threadsPerBlock>>>(deviceMask, kernelImage.cols,
                                                                        deviceUniqueMask,
                                                                        distance);
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(deviceUniqueMask));
    CHECK_CUDA(hipMemcpy(result.data, deviceMask, kernelImage.total() * sizeof(uint), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(deviceMask));

    cv::Mat croppedImage = cv::Mat(result, cv::Rect(widthPadding, heightPadding, result.cols - widthPadding, result.rows - heightPadding));
    croppedImage.copyTo(result);

    return result;
}

void PLImg::cuda::raw::filters::CUDAmedianFilter(cv::Mat& image, cv::Mat& result) {
    float* deviceImage, *deviceResult;
    int nSrcStep, nResStep;
    int2 subImageDims;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;
    // Allocate GPU memory for the original image and its result
    CHECK_CUDA(hipMalloc((void **) &deviceImage, image.total() * image.elemSize()));
    // Length of columns
    nSrcStep = image.cols;

    CHECK_CUDA(hipMalloc((void **) &deviceResult, image.total() * image.elemSize()));
    // Length of columns
    nResStep = result.cols;

    // Copy image from CPU to GPU
    CHECK_CUDA(hipMemcpy(deviceImage, image.data, image.total() * image.elemSize(), hipMemcpyHostToDevice));

    // Apply median filter
    subImageDims = {result.cols, result.rows};
    threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
    numBlocks = dim3(ceil(float(subImageDims.x) / threadsPerBlock.x), ceil(float(subImageDims.y) / threadsPerBlock.y));
    // Run median filter
    medianFilterKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                       deviceResult, nResStep,
                                                       subImageDims);

    // Copy result from GPU back to CPU
    CHECK_CUDA(hipMemcpy(result.data, deviceResult, image.total() * image.elemSize(), hipMemcpyDeviceToHost));

    // Free reserved memory
    CHECK_CUDA(hipFree(deviceImage));
    CHECK_CUDA(hipFree(deviceResult));
    CHECK_CUDA(hipDeviceSynchronize());
}

void PLImg::cuda::raw::filters::CUDAmedianFilterMasked(cv::Mat& image, cv::Mat& mask, cv::Mat& result) {
    float* deviceImage, *deviceResult;
    uchar* deviceMask;
    ulong nSrcStep, nMaskStep, nResStep;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;
    int2 subImageDims;

    // Allocate GPU memory for the original image, mask and its result
    CHECK_CUDA(hipMalloc((void **) &deviceImage, image.total() * image.elemSize()));
    // Length of columns
    nSrcStep = image.cols;

    CHECK_CUDA(hipMalloc((void **) &deviceMask, mask.total() * mask.elemSize()));
    // Length of columns
    nMaskStep = mask.cols;

    CHECK_CUDA(hipMalloc((void **) &deviceResult, image.total() * image.elemSize()));
    // Length of columns
    nResStep = result.cols;

    // Copy image from CPU to GPU
    CHECK_CUDA(hipMemcpy(deviceImage, image.data, image.total() * image.elemSize(), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(deviceMask, mask.data, mask.total() * mask.elemSize(), hipMemcpyHostToDevice));

    // Apply median filter
    subImageDims = {image.cols, image.rows};
    threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
    numBlocks = dim3(ceil(float(subImageDims.x) / threadsPerBlock.x), ceil(float(subImageDims.y) / threadsPerBlock.y));
    // Run median filter
    medianFilterMaskedKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                             deviceResult, nResStep,
                                                             deviceMask, nMaskStep,
                                                             subImageDims);

    CHECK_CUDA(hipMemcpy(result.data, deviceResult, image.total() * image.elemSize(), hipMemcpyDeviceToHost));

    // Free reserved memory
    CHECK_CUDA(hipFree(deviceImage));
    CHECK_CUDA(hipFree(deviceResult));
    CHECK_CUDA(hipFree(deviceMask));
    CHECK_CUDA(hipDeviceSynchronize());
}

cv::Mat PLImg::cuda::raw::CUDAhistogram(const cv::Mat &image, float minLabel, float maxLabel, uint numBins) {
    float* deviceImage;
    uint* deviceHistogram;

    CHECK_CUDA(hipMalloc(&deviceImage, image.total() * sizeof(float)));
    CHECK_CUDA(hipMemcpy(deviceImage, image.data, image.total() * sizeof(float), hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&deviceHistogram, numBins * sizeof(uint)));
    CHECK_CUDA(hipMemset(deviceHistogram, 0, numBins * sizeof(uint)));

    dim3 threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
    dim3 numBlocks = dim3(ceil(float(image.cols) / threadsPerBlock.x), ceil(float(image.rows) / threadsPerBlock.y));

    cv::Mat hostHistogram(numBins, 1, CV_32SC1);
    if(numBins * sizeof(uint) < 49152) {
        histogramSharedMem<<<numBlocks, threadsPerBlock, numBins * sizeof(uint)>>>
        (deviceImage, image.cols, image.rows, deviceHistogram, minLabel, maxLabel, numBins);
    } else {
        histogram<<<numBlocks, threadsPerBlock>>>
        (deviceImage, image.cols, image.rows, deviceHistogram, minLabel, maxLabel, numBins);
    }
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(hostHistogram.data, deviceHistogram, numBins * sizeof(uint), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(deviceHistogram));
    CHECK_CUDA(hipFree(deviceImage));

    return hostHistogram;
}

