#include "hip/hip_runtime.h"
/*
    MIT License

    Copyright (c) 2021 Forschungszentrum Jülich / Jan André Reuter.

    Permission is hereby granted, free of charge, to any person obtaining a copy
    of this software and associated documentation files (the "Software"), to deal
    in the Software without restriction, including without limitation the rights
    to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
    copies of the Software, and to permit persons to whom the Software is
    furnished to do so, subject to the following conditions:

    The above copyright notice and this permission notice shall be included in all
    copies or substantial portions of the Software.

    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
    AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
    SOFTWARE.
 */

#include "cuda/cuda_toolbox.h"

cv::Mat PLImg::cuda::raw::labeling::CUDAConnectedComponents(const cv::Mat& image, uint* maxLabelNumber) {
    // Prepare image for CUDA kernel
    cv::Mat kernelImage;
    // 1. Convert it to 8 bit unsigned integer values
    image.convertTo(kernelImage, CV_8UC1);
    // 2. Check if the image needs padding to allow the execution of our CUDA kernel
    uint heightPadding = CUDA_KERNEL_NUM_THREADS - kernelImage.rows % CUDA_KERNEL_NUM_THREADS;
    uint widthPadding = CUDA_KERNEL_NUM_THREADS - kernelImage.cols % CUDA_KERNEL_NUM_THREADS;
    cv::copyMakeBorder(kernelImage, kernelImage, heightPadding, 0, widthPadding, 0, cv::BORDER_CONSTANT, 0);

    // Create output resulting image for our needs
    cv::Mat result = cv::Mat(kernelImage.rows, kernelImage.cols, CV_32SC1);

    uchar* deviceImage;
    uint* deviceMask;
    bool* deviceChangeOccured;
    bool changeOccured;

    CHECK_CUDA(hipMalloc(&deviceImage, kernelImage.total() * sizeof(uchar)));
    CHECK_CUDA(hipMemcpy(deviceImage, kernelImage.data, kernelImage.total() * sizeof(uchar), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMalloc(&deviceMask, kernelImage.total() * sizeof(uint)));
    CHECK_CUDA(hipMalloc(&deviceChangeOccured, sizeof(bool)));

    dim3 threadsPerBlock, numBlocks;
    threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
    numBlocks = dim3(ceil(float(kernelImage.cols) / threadsPerBlock.x), ceil(float(kernelImage.rows) / threadsPerBlock.y));

    std::cout << numBlocks.x * threadsPerBlock.x << " " << numBlocks.y * threadsPerBlock.y << std::endl;
    std::cout << kernelImage.cols << " " << kernelImage.rows << std::endl;
    std::cout << int(heightPadding) << " " << int(widthPadding) << std::endl;

    connectedComponentsInitializeMask<<<numBlocks, threadsPerBlock>>>(deviceImage, kernelImage.cols, deviceMask, kernelImage.cols, kernelImage.cols);
    CHECK_CUDA(hipFree(deviceImage));
    do {
        CHECK_CUDA(hipMemset(deviceChangeOccured, false, sizeof(bool)));
        connectedComponentsIteration<<<numBlocks, threadsPerBlock>>>(deviceMask, kernelImage.cols, {kernelImage.cols, kernelImage.rows},
                                                                     deviceChangeOccured);
        CHECK_CUDA(hipMemcpy(&changeOccured, deviceChangeOccured, sizeof(bool), hipMemcpyDeviceToHost));
    } while(changeOccured);
    CHECK_CUDA(hipFree(deviceChangeOccured));

    uint* deviceUniqueMask;
    CHECK_CUDA(hipMalloc(&deviceUniqueMask, kernelImage.total() * sizeof(uint)));
    CHECK_CUDA(hipMemcpy(deviceUniqueMask, deviceMask, kernelImage.total() * sizeof(uint), hipMemcpyDeviceToDevice));
    thrust::sort(thrust::device, deviceUniqueMask, deviceUniqueMask + kernelImage.total());
    uint* deviceMaxUniqueLabel = thrust::unique(thrust::device, deviceUniqueMask, deviceUniqueMask + kernelImage.total());

    uint distance = thrust::distance(deviceUniqueMask, deviceMaxUniqueLabel);
    connectedComponentsReduceComponents<<<numBlocks, threadsPerBlock>>>(deviceMask, kernelImage.cols,
                                                                        deviceUniqueMask,
                                                                        distance);
    CHECK_CUDA(hipFree(deviceUniqueMask));

    uint* deviceMaxLabel = thrust::max_element(thrust::device, deviceMask, deviceMask + kernelImage.total());
    CHECK_CUDA(hipMemcpy(maxLabelNumber, deviceMaxLabel, sizeof(uint), hipMemcpyDeviceToHost));

    // Copy result from GPU back to CPU
    CHECK_CUDA(hipMemcpy(result.data, deviceMask, kernelImage.total() * sizeof(uint), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(deviceMask));

    CHECK_CUDA(hipDeviceSynchronize());

    cv::Mat croppedImage = cv::Mat(result, cv::Rect(widthPadding, heightPadding, result.cols - widthPadding, result.rows - heightPadding));
    croppedImage.copyTo(result);

    return result;
}

std::shared_ptr<cv::Mat> PLImg::cuda::raw::filters::CUDAmedianFilter(const std::shared_ptr<cv::Mat>& image) {
    // Create a result image with the same dimensions as our input image
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());
    // Expand borders of input image inplace to ensure that the median algorithm can run correcly
    cv::copyMakeBorder(*image, *image, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

    // The image might be too large to be saved completely in the video memory.
    // Therefore chunks will be used if the amount of memory is too small.
    uint numberOfChunks = 1;
    // Check the free video memory
    ulong freeMem;
    CHECK_CUDA(hipMemGetInfo(&freeMem, nullptr));
    // If the total free memory is smaller than the estimated amount of memory, calculate the number of chunks
    // with the power of four (1, 4, 16, 256, 1024, ...)
    if(double(image->total()) * image->elemSize() * 2.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 2.1 / double(freeMem)) / log(4))));
    }
    // Each dimensions will get the same number of chunks. Calculate them by using the square root.
    uint chunksPerDim = fmax(1, sqrtf(numberOfChunks));

    float* deviceImage, *deviceResult;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nResStep;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;

    // We've increased the image dimensions earlier. Save the original image dimensions for further calculations.
    int2 realImageDims = {image->cols - 2 * MEDIAN_KERNEL_SIZE, image->rows - 2 * MEDIAN_KERNEL_SIZE};
    int2 subImageDims;

    cv::Mat subImage, subResult, croppedImage;
    // For each chunk
    for(uint it = 0; it < numberOfChunks; ++it) {
        std::cout << "\rCurrent chunk: " << it+1 << "/" << numberOfChunks;
        std::flush(std::cout);
        // Calculate image boarders
        xMin = (it % chunksPerDim) * realImageDims.x / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * realImageDims.x / chunksPerDim, realImageDims.x);
        yMin = (it / chunksPerDim) * realImageDims.y / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * realImageDims.y / chunksPerDim, realImageDims.y);

        // Get chunk of our image and result. Apply padding to the result to ensure that the median filter will run correctly.
        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);
        cv::copyMakeBorder(subResult, subResult, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

        // Allocate GPU memory for the original image and its result
        CHECK_CUDA(hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize()));
        // Length of columns
        nSrcStep = subImage.cols;

        CHECK_CUDA(hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize()));
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        CHECK_CUDA(hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice));

        // Apply median filter
        subImageDims = {subImage.cols, subImage.rows};
        threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
        numBlocks = dim3(ceil(float(subImageDims.x) / threadsPerBlock.x), ceil(float(subImageDims.y) / threadsPerBlock.y));
        // Run median filter
        medianFilterKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                           deviceResult, nResStep,
                                                           subImageDims);

        // Copy result from GPU back to CPU
        CHECK_CUDA(hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost));

        // Free reserved memory
        CHECK_CUDA(hipFree(deviceImage));
        CHECK_CUDA(hipFree(deviceResult));
        CHECK_CUDA(hipDeviceSynchronize());

        // Calculate the range where the median filter was applied and where the chunk will be placed.
        cv::Rect srcRect = cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, xMax - xMin, yMax - yMin);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);
        subResult(srcRect).copyTo(result(dstRect));
    }
    // Fix output after \r
    std::cout << std::endl;
    // Revert the padding of the original image
    croppedImage = cv::Mat(*image, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, image->cols - 2*MEDIAN_KERNEL_SIZE, image->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*image);

    // Return resulting median filtered image
    return std::make_shared<cv::Mat>(result);
}

std::shared_ptr<cv::Mat> PLImg::cuda::raw::filters::CUDAmedianFilterMasked(const std::shared_ptr<cv::Mat>& image, const std::shared_ptr<cv::Mat>& mask) {
    // Copy the result back to the CPU
    cv::Mat result = cv::Mat(image->rows, image->cols, image->type());
    cv::copyMakeBorder(*image, *image, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);
    cv::copyMakeBorder(*mask, *mask, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

    // The image might be too large to be saved completely in the video memory.
    // Therefore chunks will be used if the amount of memory is too small.
    uint numberOfChunks = 1;
    ulong freeMem;
    // Check the free video memory
    CHECK_CUDA(hipMemGetInfo(&freeMem, nullptr));
    // If the total free memory is smaller than the estimated amount of memory, calculate the number of chunks
    // with the power of four (1, 4, 16, 256, 1024, ...)
    if(double(image->total()) * image->elemSize() * 3.1 > double(freeMem)) {
        numberOfChunks = fmax(1, pow(4.0, ceil(log(image->total() * image->elemSize() * 3.1 / double(freeMem)) / log(4))));
    }
    // Each dimensions will get the same number of chunks. Calculate them by using the square root.
    uint chunksPerDim = fmax(1, sqrtf(numberOfChunks));

    float* deviceImage, *deviceResult;
    uchar* deviceMask;
    uint xMin, xMax, yMin, yMax;
    ulong nSrcStep, nMaskStep, nResStep;
    // Apply median filter
    // Calculate offsets for image and result. Starting at the edge would result in errors because we would
    // go out of bounds.
    dim3 threadsPerBlock, numBlocks;

    // We've increased the image dimensions earlier. Save the original image dimensions for further calculations.
    int2 realImageDims = {image->cols - 2 * MEDIAN_KERNEL_SIZE, image->rows - 2 * MEDIAN_KERNEL_SIZE};
    int2 subImageDims;

    cv::Mat subImage, subMask, subResult, croppedImage;
    for(uint it = 0; it < numberOfChunks; ++it) {
        std::cout << "\rCurrent chunk: " << it+1 << "/" << numberOfChunks;
        std::flush(std::cout);
        // Calculate image boarders
        xMin = (it % chunksPerDim) * realImageDims.x / chunksPerDim;
        xMax = fmin((it % chunksPerDim + 1) * realImageDims.x / chunksPerDim, realImageDims.x);
        yMin = (it / chunksPerDim) * realImageDims.y / chunksPerDim;
        yMax = fmin((it / chunksPerDim + 1) * realImageDims.y / chunksPerDim, realImageDims.y);

        // Get chunk of our image, mask and result. Apply padding to the result to ensure that the median filter will run correctly.
        croppedImage = cv::Mat(*image, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subImage);
        croppedImage = cv::Mat(*mask, cv::Rect(xMin, yMin, xMax - xMin + 2 * MEDIAN_KERNEL_SIZE, yMax - yMin + 2 * MEDIAN_KERNEL_SIZE));
        croppedImage.copyTo(subMask);
        croppedImage = cv::Mat(result, cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin));
        croppedImage.copyTo(subResult);
        cv::copyMakeBorder(subResult, subResult, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, cv::BORDER_REPLICATE);

        // Allocate GPU memory for the original image, mask and its result
        CHECK_CUDA(hipMalloc((void **) &deviceImage, subImage.total() * subImage.elemSize()));
        // Length of columns
        nSrcStep = subImage.cols;

        CHECK_CUDA(hipMalloc((void **) &deviceMask, subMask.total() * subMask.elemSize()));
        // Length of columns
        nMaskStep = subMask.cols;

        CHECK_CUDA(hipMalloc((void **) &deviceResult, subImage.total() * subImage.elemSize()));
        // Length of columns
        nResStep = subResult.cols;

        // Copy image from CPU to GPU
        CHECK_CUDA(hipMemcpy(deviceImage, subImage.data, subImage.total() * subImage.elemSize(), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(deviceMask, subMask.data, subMask.total() * subMask.elemSize(), hipMemcpyHostToDevice));

        // Apply median filter
        subImageDims = {subImage.cols, subImage.rows};
        threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
        numBlocks = dim3(ceil(float(subImageDims.x) / threadsPerBlock.x), ceil(float(subImageDims.y) / threadsPerBlock.y));
        // Run median filter
        medianFilterMaskedKernel<<<numBlocks, threadsPerBlock>>>(deviceImage, nSrcStep,
                                                                 deviceResult, nResStep,
                                                                 deviceMask, nMaskStep,
                                                                 subImageDims);

        CHECK_CUDA(hipMemcpy(subResult.data, deviceResult, subImage.total() * subImage.elemSize(), hipMemcpyDeviceToHost));

        // Free reserved memory
        CHECK_CUDA(hipFree(deviceImage));
        CHECK_CUDA(hipFree(deviceResult));
        CHECK_CUDA(hipFree(deviceMask));
        CHECK_CUDA(hipDeviceSynchronize());

        cv::Rect srcRect = cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, subResult.cols - 2*MEDIAN_KERNEL_SIZE, subResult.rows - 2*MEDIAN_KERNEL_SIZE);
        cv::Rect dstRect = cv::Rect(xMin, yMin, xMax - xMin, yMax - yMin);

        subResult(srcRect).copyTo(result(dstRect));
    }
    // Fix output after \r
    std::cout << std::endl;

    croppedImage = cv::Mat(*image, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, image->cols - 2*MEDIAN_KERNEL_SIZE, image->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*image);
    croppedImage = cv::Mat(*mask, cv::Rect(MEDIAN_KERNEL_SIZE, MEDIAN_KERNEL_SIZE, mask->cols - 2*MEDIAN_KERNEL_SIZE, mask->rows - 2*MEDIAN_KERNEL_SIZE));
    croppedImage.copyTo(*mask);
    return std::make_shared<cv::Mat>(result);
}

cv::Mat PLImg::cuda::raw::CUDAhistogram(const cv::Mat &image, uint *minLabel, uint *maxLabel) {
    uint* deviceImage;
    uint* deviceHistogram;

    CHECK_CUDA(hipMalloc(&deviceImage, image.total() * sizeof(uint)));
    CHECK_CUDA(hipMemcpy(deviceImage, image.data, image.total() * sizeof(uint), hipMemcpyHostToDevice));

    uint* deviceMinLabel = nullptr;
    uint *deviceMaxLabel = nullptr;
    deviceMaxLabel = thrust::max_element(thrust::device, deviceImage, deviceImage + image.total());
    deviceMinLabel = thrust::min_element(thrust::device, deviceImage, deviceImage + image.total());
    CHECK_CUDA(hipMemcpy(minLabel, deviceMinLabel, sizeof(uint), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(maxLabel, deviceMaxLabel, sizeof(uint), hipMemcpyDeviceToHost));

    std::cout << "Min Label = " << *minLabel << ", Max Label = " << *maxLabel << std::endl;
    std::cout << (*maxLabel - *minLabel) << std::endl;

    CHECK_CUDA(hipMalloc(&deviceHistogram, (*maxLabel - *minLabel) * sizeof(uint)));
    CHECK_CUDA(hipMemset(deviceHistogram, 0, (*maxLabel - *minLabel) * sizeof(uint)));

    dim3 threadsPerBlock = dim3(CUDA_KERNEL_NUM_THREADS, CUDA_KERNEL_NUM_THREADS);
    dim3 numBlocks = dim3(ceil(float(image.cols) / threadsPerBlock.x), ceil(float(image.rows) / threadsPerBlock.y));

    histogram<<<numBlocks, threadsPerBlock>>>(deviceImage, image.cols, image.rows, deviceHistogram, *minLabel, *maxLabel);
    CHECK_CUDA(hipFree(deviceImage));

    cv::Mat hostHistogram(*maxLabel - *minLabel, 1, CV_32SC1);

    CHECK_CUDA(hipMemcpy(hostHistogram.data, deviceHistogram, (*maxLabel - *minLabel) * sizeof(uint), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(deviceHistogram));

    return hostHistogram;
}

